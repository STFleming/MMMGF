
#include <hip/hip_runtime.h>
/* myapp_m1.cu ---- Brittle */
/* GPU version */

__global__ void kernel(int *a, int *b) {
  a[threadIdx.x] += b[threadIdx.x];
}

extern "C" {

void cukernel(int *a, int *b, int size) {
  kernel <<<1, size>>>(a, b);
}

}
