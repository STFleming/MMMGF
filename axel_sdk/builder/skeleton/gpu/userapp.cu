/* userapp.cu
 * by Brittle 2009
 *
 * Template for CUDA programming on AXEL cluster
 */


#include <hip/hip_runtime.h>
#include <stdio.h>

#define N 1000
#define tpb 256
#define SIZE N*sizeof(float)

__global__ void kernel(float *A, float *B, float *C) {
  int i = blockIdx.x * 256 + threadIdx.x;
  if (i < N) // check since some threads may be created uselessly
    C[i] = A[i] + B[i];
}

int main(void) {
  float a[N], b[N], c[N];
  float *A, *B, *C;
  int i;

  for (i=0; i<N; i++) {
    a[i] = i;
    b[i] = 2 * i;
    c[i] = 0;
  }

  hipMalloc((void **)&A, SIZE);
  hipMalloc((void **)&B, SIZE);
  hipMalloc((void **)&C, SIZE);

  hipMemcpy(A, a, SIZE, hipMemcpyHostToDevice);
  hipMemcpy(B, b, SIZE, hipMemcpyHostToDevice);

  kernel<<<(N+tpb-1)/tpb, tpb>>>(A, B, C);

  hipMemcpy(c, C, SIZE, hipMemcpyDeviceToHost);

  hipFree(A);
  hipFree(B);
  hipFree(C);

  for (i=0; i<N; i++) {
    if (i%10 == 0) printf("\n");
    printf("%6.0f ", c[i]);
  }
  printf("\n");

  return 0;
}
